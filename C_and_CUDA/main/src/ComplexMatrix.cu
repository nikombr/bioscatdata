#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <string.h>
extern "C" {
#include "../lib/ComplexMatrix.h"
using namespace std;

ComplexMatrix::ComplexMatrix(int rows, int cols) {

    // Save input parameters
    this->rows       = rows;
    this->cols       = cols;
    this->host       = true;
    this->device     = true;

    // Allocate vectors on host
    hipHostMalloc((void **) &real_h, rows*cols*sizeof(double));
    hipHostMalloc((void **) &imag_h, rows*cols*sizeof(double));

    // Allocate vectors on device
    hipMalloc((void **) &real_d, rows*cols*sizeof(double));
    hipMalloc((void **) &imag_d, rows*cols*sizeof(double));
}

ComplexMatrix::ComplexMatrix(int rows, int cols, bool host, bool device) {

    // Save input parameters
    this->rows       = rows;
    this->cols       = cols;
    this->host       = host;
    this->device     = device;

    if (host) {
        // Allocate vectors on host
        hipHostMalloc((void **) &real_h, rows*cols*sizeof(double));
        hipHostMalloc((void **) &imag_h, rows*cols*sizeof(double));
    }

    if (device) {
        // Allocate vectors on device
        hipMalloc((void **) &real_d, rows*cols*sizeof(double));
        hipMalloc((void **) &imag_d, rows*cols*sizeof(double));
    }
}

ComplexMatrix::ComplexMatrix(int rows) {

    // Save input parameters
    this->rows       = rows;
    this->cols       = 1;
    this->host       = true;
    this->device     = true;

    // Allocate vectors on host
    hipHostMalloc((void **) &real_h, rows*cols*sizeof(double));
    hipHostMalloc((void **) &imag_h, rows*cols*sizeof(double));

    // Allocate vectors on device
    hipMalloc((void **) &real_d, rows*cols*sizeof(double));
    hipMalloc((void **) &imag_d, rows*cols*sizeof(double));
}

ComplexMatrix::ComplexMatrix(int rows, bool host, bool device) {

    // Save input parameters
    this->rows       = rows;
    this->cols       = 1;
    this->host       = host;
    this->device     = device;

    if (host) {
        // Allocate vectors on host
        hipHostMalloc((void **) &real_h, rows*cols*sizeof(double));
        hipHostMalloc((void **) &imag_h, rows*cols*sizeof(double));
    }

    if (device) {
        // Allocate vectors on device
        hipMalloc((void **) &real_d, rows*cols*sizeof(double));
        hipMalloc((void **) &imag_d, rows*cols*sizeof(double));
    }
}

void ComplexMatrix::free() {
    hipError_t err;
    if (host) {
        // Free on host
        err = hipHostFree(real_h);

        if (err != hipSuccess) {
            std::cerr << "Failed to free memory on host: " << hipGetErrorString(err) << std::endl;
        }
        err = hipHostFree(imag_h);

        if (err != hipSuccess) {
            std::cerr << "Failed to free memory on host: " << hipGetErrorString(err) << std::endl;
        }
    }
    if (device) {
        // Free on device
        err = hipFree(real_d);
        if (err != hipSuccess) {
            std::cerr << "Failed to free memory on device: " << hipGetErrorString(err) << std::endl;
        }
        err = hipFree(imag_d);
        if (err != hipSuccess) {
            std::cerr << "Failed to free memory on device: " << hipGetErrorString(err) << std::endl;
        }
    }

}

void ComplexMatrix::toHost() {
    if (host && device) {
        // Send from device to host
        hipMemcpy(real_h, real_d, rows * cols * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(imag_h, imag_d, rows * cols * sizeof(double), hipMemcpyDeviceToHost);
    }

}

void ComplexMatrix::toDevice() {

    if (host && device) {
        // Send from host to device
        hipMemcpy(real_d, real_h, rows * cols * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(imag_d, imag_h, rows * cols * sizeof(double), hipMemcpyHostToDevice);
    }
    
}

void ComplexMatrix::setHostRealValue(int r, double val) {
    real_h[r] = val;
}

void ComplexMatrix::setHostRealValue(int r, int c, double val) {
    real_h[r*cols + c] = val;
}

double ComplexMatrix::getHostRealValue(int r) {
    return real_h[r];
}

double ComplexMatrix::getHostRealValue(int r, int c) {
    return real_h[r*cols + c];
}


void ComplexMatrix::setHostImagValue(int r, double val) {
    imag_h[r] = val;
}

void ComplexMatrix::setHostImagValue(int r, int c, double val) {
    imag_h[r*cols + c] = val;
}


double ComplexMatrix::getHostImagValue(int r) {
    return imag_h[r];
}

double ComplexMatrix::getHostImagValue(int r, int c) {
    return imag_h[r*cols + c];
}


void ComplexMatrix::dumpResult(const char * filename) {
    if (cols == 1) {
        FILE *file;
        file = fopen(filename, "w");
        if (file == NULL) {
            perror("Error opening file");
            return;
        }
        for (int r = 0; r < rows; r++) {
            fprintf(file, "%e\t%e\n", getHostRealValue(r), getHostImagValue(r));
        }
        fclose(file);
    }
    else printf("We do not support saving complex matrices with several columns.\n");
}


}