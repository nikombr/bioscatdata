#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hipsolver.h>
extern "C" {
#include "../../lib/GP/GaussianProcess.h"

// Need to transpose manually as cusolver does not seem to have implemented this yet
__global__ void transposeMatrix(double ** input, double ** output, int n) {
    int r = threadIdx.x + blockIdx.x * blockDim.x;
    int c = threadIdx.y + blockIdx.y * blockDim.y;
    if (r < n && c < n) {
        output[c][r] = input[r][c];
    }
}

__global__ void copyMatrix(double ** input, double ** output, int n) {
    int r = threadIdx.x + blockIdx.x * blockDim.x;
    int c = threadIdx.y + blockIdx.y * blockDim.y;
    if (r < n && c < n) {
        output[r][c] = input[r][c];
    }
}


void GaussianProcess::compute_inverse() {


    if (device) {

        hipsolverStatus_t status;

        // Move the cholesky factorized covariance matrix to avoid overwrites
        dim3 dimBlock(32,32);
        dim3 dimGrid((n + dimBlock.x - 1)/dimBlock.x, (n + dimBlock.y - 1)/dimBlock.y);
        copyMatrix<<<dimGrid, dimBlock>>>(M_d, M_inv_d, n);
        hipDeviceSynchronize();

        // Get cuSolver handle
        hipsolverHandle_t handle;
        hipsolverDnCreate(&handle);

        hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;

        // Workspace query
        int work_size = 0;
        status =  hipsolverDnDpotri_bufferSize(handle, uplo, n, M_inv_log, n, &work_size);

        // Allocate workspace
        double * workspace_d;
        hipMalloc((void**) &workspace_d, work_size * sizeof(double));
        int * info_d;
        hipMalloc((void**)&info_d, sizeof(int));

        // Get inverse
        status = hipsolverDnDpotri(handle, uplo, n, M_inv_log, n, workspace_d, work_size, info_d);
        hipDeviceSynchronize();
        
        // Check if successful
        int info_h = 0;
        hipMemcpy(&info_h, info_d, sizeof(int), hipMemcpyDeviceToHost);
        if (info_h != 0) {
            printf("Computing inverse failed with info = %d\n", info_h);
        }
        

        hipFree(workspace_d);
        hipFree(info_d);
        hipsolverDnDestroy(handle);
        hipDeviceSynchronize();


    }
    else {
        printf("Error: Not implemented on host!\n"); // Fine as we probably won't use it on the host
    }


}

}