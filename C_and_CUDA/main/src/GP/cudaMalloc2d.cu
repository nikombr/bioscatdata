// This file is reused from the HPC course and the advanced special course in HPC


#include <hip/hip_runtime.h>
#include <stdlib.h>
extern "C" {
void
host_malloc_2d(double ***B, int n) {

    double **p;
    double *a;

    if (n <= 0) {
        B = NULL;
        return;
    }

    hipHostMalloc(&p, n * sizeof(double *), hipHostMallocDefault);
    //double **A = malloc(n * sizeof(double *));
    
    if (p == NULL) {
        B = NULL;
        return;
    }

    hipHostMalloc(&a, n * n * sizeof(double), hipHostMallocDefault);
    //A[0] = malloc(n*n*sizeof(double));

    if (a == NULL) {
        hipHostFree(p);
        B = NULL;
        return;
    }

    for (int i = 0; i < n; i++) p[i] = a + i * n;

    *B = p;
}

void
host_free_2d(double **B) {
    hipHostFree(B[0]);
    hipHostFree(B);
}

__global__ void mallocLoops(double**p, double *a, int n) {

    for (int i = 0; i < n; i++) p[i] = a + i * n;

}

void
device_malloc_2d(double ***B,double **b,int n) {

    double **p;
    double *a;

    if (n <= 0) {
        B = NULL;
        return;
    }

    hipMalloc(&p, n * sizeof(double *));
    
    if (p == NULL) {
        B = NULL;
        return;
    }

    hipMalloc(&a, n * n * sizeof(double));


    if (a == NULL) {
        hipFree(p);
        B = NULL;
        return;
    }

    mallocLoops<<<1,1>>>(p, a, n);
    hipDeviceSynchronize();

    *B = p;
    *b = a;
}

void device_free_2d(double **B,double *b) {
    hipFree(b);
    hipFree(B);
}
}