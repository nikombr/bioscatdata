#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
//#include <hip/hip_runtime_api.h>
#include <iostream>
#include <string.h>
#include <omp.h>
#include "../../lib/Segment.h"
#include "../../lib/RealMatrix.h"
extern "C" {
using namespace std;

#define cudaCheckError() {                                          \
    hipError_t e = hipGetLastError();                             \
    if (e != hipSuccess) {                                         \
        printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        exit(EXIT_FAILURE);                                         \
    }                                                               \
}

__global__ void computeScatteredSubFieldsKernel(ComplexMatrix C, ComplexMatrix F1, ComplexMatrix F2, ComplexMatrix F3, ComplexMatrix F1_matrix, ComplexMatrix F2_matrix, ComplexMatrix F3_matrix, int rows, int cols) {
    int k = threadIdx.x + blockIdx.x * blockDim.x;
    if (k < rows) {
        
        double f1_real, f1_imag, f2_real, f2_imag, f3_real, f3_imag;
        f1_real = 0.0; f2_real = 0.0; f3_real = 0.0;
        f1_imag = 0.0; f2_imag = 0.0; f3_imag = 0.0;
        
        for (int j = 0; j < cols; j++) {
            // Get factor
            double C_real = C.getDeviceRealValue(j);
            double C_imag = C.getDeviceImagValue(j);

            // Computing real values
            f1_real += F1_matrix.getDeviceRealValue(k,j) * C_real;
            f1_real -= F1_matrix.getDeviceImagValue(k,j) * C_imag;
            f2_real += F2_matrix.getDeviceRealValue(k,j) * C_real;
            f2_real -= F2_matrix.getDeviceImagValue(k,j) * C_imag;
            f3_real += F3_matrix.getDeviceRealValue(k,j) * C_real;
            f3_real -= F3_matrix.getDeviceImagValue(k,j) * C_imag;

            // Computing imagninary values
            f1_imag += F1_matrix.getDeviceRealValue(k,j) * C_imag;
            f1_imag += F1_matrix.getDeviceImagValue(k,j) * C_real;
            f2_imag += F2_matrix.getDeviceRealValue(k,j) * C_imag;
            f2_imag += F2_matrix.getDeviceImagValue(k,j) * C_real;
            f3_imag += F3_matrix.getDeviceRealValue(k,j) * C_imag;
            f3_imag += F3_matrix.getDeviceImagValue(k,j) * C_real;
    
        }

        F1.setDeviceRealValue(k, f1_real);
        F2.setDeviceRealValue(k, f2_real);
        F3.setDeviceRealValue(k, f3_real);
        F1.setDeviceImagValue(k, f1_imag);
        F2.setDeviceImagValue(k, f2_imag);
        F3.setDeviceImagValue(k, f3_imag);

        //if (k == 0) printf("E_scat = %f\n",C.getDeviceRealValue(0));
    }
    
}

void Segment::computeScatteredSubFields() {

    int rows = n_obs;
    int cols = E_scat_matrix.cols;

    if (deviceComputation) {
        //E_scat_matrix.toDevice();
        //H_scat_matrix.toDevice();
        double start = omp_get_wtime();
        // Blocks and threads
        dim3 dimBlock(256);
        dim3 dimGrid((rows + dimBlock.x - 1)/dimBlock.x);
        if (polarisation == 1) {
            computeScatteredSubFieldsKernel<<<dimGrid, dimBlock>>>(C, E_scat.z, H_scat.x, H_scat.y, E_scat_matrix.z, H_scat_matrix.x, H_scat_matrix.y, rows, cols);
        }
        else if (polarisation == 2) {
            computeScatteredSubFieldsKernel<<<dimGrid, dimBlock>>>(C, H_scat.z, E_scat.x, E_scat.y, H_scat_matrix.z, E_scat_matrix.x, E_scat_matrix.y, rows, cols);
        }
        cudaCheckError();

        

        hipDeviceSynchronize();
        double end = omp_get_wtime();
        //printf("time = %e\n",end-start);
        //E_scat.toHost();
        //H_scat.toHost();

    }
    else {
        if (polarisation == 1) {
            #pragma omp parallel for
            for (int k = 0; k < rows; k++) {
                double Ez_real, Ez_imag, Hx_real, Hx_imag, Hy_real, Hy_imag;
                Ez_real = 0.0; Hx_real = 0.0; Hy_real = 0.0;
                Ez_imag = 0.0; Hx_imag = 0.0; Hy_imag = 0.0;
                
                for (int j = 0; j < cols; j++) {
                    // Get factor
                    double C_real = C.getHostRealValue(j);
                    double C_imag = C.getHostImagValue(j);

                    // Computing real values
                    Ez_real += E_scat_matrix.z.getHostRealValue(k,j) * C_real;
                    Ez_real -= E_scat_matrix.z.getHostImagValue(k,j) * C_imag;
                    Hx_real += H_scat_matrix.x.getHostRealValue(k,j) * C_real;
                    Hx_real -= H_scat_matrix.x.getHostImagValue(k,j) * C_imag;
                    Hy_real += H_scat_matrix.y.getHostRealValue(k,j) * C_real;
                    Hy_real -= H_scat_matrix.y.getHostImagValue(k,j) * C_imag;

                    // Computing imagninary values
                    Ez_imag += E_scat_matrix.z.getHostRealValue(k,j) * C_imag;
                    Ez_imag += E_scat_matrix.z.getHostImagValue(k,j) * C_real;
                    Hx_imag += H_scat_matrix.x.getHostRealValue(k,j) * C_imag;
                    Hx_imag += H_scat_matrix.x.getHostImagValue(k,j) * C_real;
                    Hy_imag += H_scat_matrix.y.getHostRealValue(k,j) * C_imag;
                    Hy_imag += H_scat_matrix.y.getHostImagValue(k,j) * C_real;
            
                }

                E_scat.z.setHostRealValue(k, Ez_real);
                H_scat.x.setHostRealValue(k, Hx_real);
                H_scat.y.setHostRealValue(k, Hy_real);
                E_scat.z.setHostImagValue(k, Ez_imag);
                H_scat.x.setHostImagValue(k, Hx_imag);
                H_scat.y.setHostImagValue(k, Hy_imag);
            }

        }
        else if (polarisation == 2) {
            #pragma omp parallel for
            for (int k = 0; k < rows; k++) {
                double Hz_real, Hz_imag, Ex_real, Ex_imag, Ey_real, Ey_imag, C_real, C_imag;
                Hz_real = 0.0; Ex_real = 0.0; Ey_real = 0.0;
                Hz_imag = 0.0; Ex_imag = 0.0; Ey_imag = 0.0;
                for (int j = 0; j < cols; j++) {
                    // Get factor
                    C_real = C.getHostRealValue(j);
                    C_imag = C.getHostImagValue(j);

                    // Computing real values
                    Hz_real += H_scat_matrix.z.getHostRealValue(k,j) * C_real;
                    Hz_real -= H_scat_matrix.z.getHostImagValue(k,j) * C_imag;
                    Ex_real += E_scat_matrix.x.getHostRealValue(k,j) * C_real;
                    Ex_real -= E_scat_matrix.x.getHostImagValue(k,j) * C_imag;
                    Ey_real += E_scat_matrix.y.getHostRealValue(k,j) * C_real;
                    Ey_real -= E_scat_matrix.y.getHostImagValue(k,j) * C_imag;

                    // Computing complex values
                    Hz_imag += H_scat_matrix.z.getHostRealValue(k,j) * C_imag;
                    Hz_imag += H_scat_matrix.z.getHostImagValue(k,j) * C_real;
                    Ex_imag += E_scat_matrix.x.getHostRealValue(k,j) * C_imag;
                    Ex_imag += E_scat_matrix.x.getHostImagValue(k,j) * C_real;
                    Ey_imag += E_scat_matrix.y.getHostRealValue(k,j) * C_imag;
                    Ey_imag += E_scat_matrix.y.getHostImagValue(k,j) * C_real;
            
                }

                H_scat.z.setHostRealValue(k, Hz_real);
                E_scat.x.setHostRealValue(k, Ex_real);
                E_scat.y.setHostRealValue(k, Ey_real);
                H_scat.z.setHostImagValue(k, Hz_imag);
                E_scat.x.setHostImagValue(k, Ex_imag);
                E_scat.y.setHostImagValue(k, Ey_imag);
            }

        }
    }
}




}