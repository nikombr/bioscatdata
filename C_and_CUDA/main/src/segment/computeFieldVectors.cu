#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <string.h>
#include "../../lib/Segment.h"
extern "C" {
#include "../../lib/RealMatrix.h"
using namespace std;

__global__ void computeFieldKernelReal(ComplexMatrix field, RealMatrix y, int n, double interior_constant, double exterior_constant) {
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    if (j < n) {
        field.setDeviceRealValue(j, exterior_constant * cos(interior_constant * y.getDeviceValue(j)));
    }
}

__global__ void computeFieldKernelImag(ComplexMatrix field, RealMatrix y, int n, double interior_constant, double exterior_constant) {
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    if (j < n) {
        field.setDeviceImagValue(j, exterior_constant * sin(interior_constant * y.getDeviceValue(j)));
    }
}


void Segment::computeIncidentFieldVectors(RealMatrix y) {
    
    int rows = y.rows;

    if (true) {
        // Blocks and threads
        dim3 dimBlock(256);
        dim3 dimGrid((rows + dimBlock.x - 1)/dimBlock.x);
        if (polarisation == 1) {
            computeFieldKernelReal<<<dimGrid, dimBlock>>>(E_inc_vector.z, y, rows, constants.k0, 1.0);
            computeFieldKernelImag<<<dimGrid, dimBlock>>>(E_inc_vector.z, y, rows, constants.k0, 1.0);
            computeFieldKernelReal<<<dimGrid, dimBlock>>>(H_inc_vector.x, y, rows, constants.k0, -1/constants.eta0);
            computeFieldKernelImag<<<dimGrid, dimBlock>>>(H_inc_vector.x, y, rows, constants.k0, -1/constants.eta0);
        }
        else if (polarisation == 2) {
            computeFieldKernelReal<<<dimGrid, dimBlock>>>(E_inc_vector.x, y, rows, constants.k0, 1.0);
            computeFieldKernelImag<<<dimGrid, dimBlock>>>(E_inc_vector.x, y, rows, constants.k0, 1.0);
            computeFieldKernelReal<<<dimGrid, dimBlock>>>(H_inc_vector.z, y, rows, constants.k0, 1/constants.eta0);
            computeFieldKernelImag<<<dimGrid, dimBlock>>>(H_inc_vector.z, y, rows, constants.k0, 1/constants.eta0);
        }

        hipDeviceSynchronize();

        //E_inc_vector.toHost();
        //H_inc_vector.toHost();

    }
    else {
        if (polarisation == 1) {
            for (int j = 0; j < rows; j++) E_inc_vector.z.setHostRealValue(j,                     cos(constants.k0 * y.getHostValue(j)));
            for (int j = 0; j < rows; j++) E_inc_vector.z.setHostImagValue(j,                     sin(constants.k0 * y.getHostValue(j)));
            for (int j = 0; j < rows; j++) H_inc_vector.x.setHostRealValue(j, -1/constants.eta0 * cos(constants.k0 * y.getHostValue(j)));
            for (int j = 0; j < rows; j++) H_inc_vector.x.setHostImagValue(j, -1/constants.eta0 * sin(constants.k0 * y.getHostValue(j)));
        }
        else {
            for (int j = 0; j < rows; j++) E_inc_vector.x.setHostRealValue(j,                    cos(constants.k0 * y.getHostValue(j)));
            for (int j = 0; j < rows; j++) E_inc_vector.x.setHostImagValue(j,                    sin(constants.k0 * y.getHostValue(j)));
            for (int j = 0; j < rows; j++) H_inc_vector.z.setHostRealValue(j, 1/constants.eta0 * cos(constants.k0 * y.getHostValue(j)));
            for (int j = 0; j < rows; j++) H_inc_vector.z.setHostImagValue(j, 1/constants.eta0 * sin(constants.k0 * y.getHostValue(j)));
        }
    }
}

void Segment::computeReflectedFieldVectors(RealMatrix y) {
    
    int rows = y.rows;

    if (deviceComputation) {
        // Blocks and threads
        dim3 dimBlock(256);
        dim3 dimGrid((rows + dimBlock.x - 1)/dimBlock.x);
        if (polarisation == 1) {
            computeFieldKernelReal<<<dimGrid, dimBlock>>>(E_ref_vector.z, y, rows, constants.k0, constants.Gamma_ref);
            computeFieldKernelImag<<<dimGrid, dimBlock>>>(E_ref_vector.z, y, rows, constants.k0, -constants.Gamma_ref);
            computeFieldKernelReal<<<dimGrid, dimBlock>>>(H_ref_vector.x, y, rows, constants.k0, 1/constants.eta0*constants.Gamma_ref);
            computeFieldKernelImag<<<dimGrid, dimBlock>>>(H_ref_vector.x, y, rows, constants.k0, -1/constants.eta0*constants.Gamma_ref);
        }
        else if (polarisation == 2) {
            computeFieldKernelReal<<<dimGrid, dimBlock>>>(E_ref_vector.x, y, rows, constants.k0, constants.Gamma_ref);
            computeFieldKernelImag<<<dimGrid, dimBlock>>>(E_ref_vector.x, y, rows, constants.k0, -constants.Gamma_ref);
            computeFieldKernelReal<<<dimGrid, dimBlock>>>(H_ref_vector.z, y, rows, constants.k0, -1/constants.eta0*constants.Gamma_ref);
            computeFieldKernelImag<<<dimGrid, dimBlock>>>(H_ref_vector.z, y, rows, constants.k0, 1/constants.eta0*constants.Gamma_ref);
        }

        hipDeviceSynchronize();

    }
    else {
        if (polarisation == 1) {
            for (int j = 0; j < rows; j++) E_ref_vector.z.setHostRealValue(j,                     constants.Gamma_ref * cos(constants.k0 * y.getHostValue(j)));
            for (int j = 0; j < rows; j++) E_ref_vector.z.setHostImagValue(j,                    -constants.Gamma_ref * sin(constants.k0 * y.getHostValue(j)));
            for (int j = 0; j < rows; j++) H_ref_vector.x.setHostRealValue(j,  1/constants.eta0 * constants.Gamma_ref * cos(constants.k0 * y.getHostValue(j)));
            for (int j = 0; j < rows; j++) H_ref_vector.x.setHostImagValue(j, -1/constants.eta0 * constants.Gamma_ref * sin(constants.k0 * y.getHostValue(j)));
        }
        else {
            for (int j = 0; j < rows; j++) E_ref_vector.x.setHostRealValue(j,                     constants.Gamma_ref * cos(constants.k0 * y.getHostValue(j)));
            for (int j = 0; j < rows; j++) E_ref_vector.x.setHostImagValue(j,                    -constants.Gamma_ref * sin(constants.k0 * y.getHostValue(j)));
            for (int j = 0; j < rows; j++) H_ref_vector.z.setHostRealValue(j, -1/constants.eta0 * constants.Gamma_ref * cos(constants.k0 * y.getHostValue(j)));
            for (int j = 0; j < rows; j++) H_ref_vector.z.setHostImagValue(j,  1/constants.eta0 * constants.Gamma_ref * sin(constants.k0 * y.getHostValue(j)));
        }
    }
}
  

}