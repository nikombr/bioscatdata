#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <string.h>
#include <omp.h>
#include "../../lib/Segment.h"
extern "C" {
#include "../../lib/RealMatrix.h"
using namespace std;

#define cudaCheckError() {                                          \
    hipError_t e = hipGetLastError();                             \
    if (e != hipSuccess) {                                         \
        printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        exit(EXIT_FAILURE);                                         \
    }                                                               \
}

__host__ __device__ double H02_real(double x) {
    // Computes real part of Hankel function of order zero and second kind
    int n = 0;
    double Jn = jn(n, x); // Compute Bessel functions of the first (Jn) 
    return Jn;
}

__host__ __device__ double H02_imag(double x) {
    // Computes imaginary part of Hankel function of order zero and second kind
    int n = 0;
    double Yn = yn(n, x); // Compute Bessel functions of the second (Yn) kind
    return -Yn;
}


__host__ __device__ double H12_real(double x) {
    // Computes real part of Hankel function of order one and second kind
    int n = 1;
    double Jn = jn(n, x); // Compute Bessel functions of the first (Jn) 
    return Jn; 
}


 __host__ __device__ double H12_imag(double x) {
    // Computes imaginary part of Hankel function of order one and second kind
    int n = 1;
    double Yn = yn(n, x); // Compute Bessel functions of the second (Yn) kind
    return -Yn;
}

__global__ void computeScatteredFieldMatricesKernelH02(ComplexMatrix F1, RealMatrix x, RealMatrix x_int, RealMatrix y, RealMatrix y_int, double constant, int rows, int cols, double k0, double Gamma_ref) {
    int r = threadIdx.x + blockIdx.x * blockDim.x;
    int c = threadIdx.y + blockIdx.y * blockDim.y;
    if (r < rows && c < cols) {
        double abs_int, abs_int_ref, xdiff, ydiff, ydiff_ref, H_real, H_imag, H_real_ref, H_imag_ref, val;

        // Get data
        xdiff       = x.getDeviceValue(r) - x_int.getDeviceValue(c);
        ydiff       = y.getDeviceValue(r) - y_int.getDeviceValue(c);
        ydiff_ref   = y.getDeviceValue(r) + y_int.getDeviceValue(c);
        abs_int     = std::sqrt(xdiff * xdiff + ydiff     * ydiff);
        abs_int_ref = std::sqrt(xdiff * xdiff + ydiff_ref * ydiff_ref);

        // Compute first Hankel functions
        H_real     = H02_real(k0 * abs_int);
        H_real_ref = H02_real(k0 * abs_int_ref);
        H_imag     = H02_imag(k0 * abs_int);
        H_imag_ref = H02_imag(k0 * abs_int_ref);
        
        
        val = H_real + Gamma_ref * H_real_ref;
        F1.setDeviceRealValue(r, c, val);
        val = H_imag + Gamma_ref * H_imag_ref;
        F1.setDeviceImagValue(r, c, val);
    }
}

__global__ void computeScatteredFieldMatricesKernelH12(ComplexMatrix F2, ComplexMatrix F3, RealMatrix x, RealMatrix x_int, RealMatrix y, RealMatrix y_int, double constant, int rows, int cols, double k0, double Gamma_ref) {
    int r = threadIdx.x + blockIdx.x * blockDim.x;
    int c = threadIdx.y + blockIdx.y * blockDim.y;
    if (r < rows && c < cols) {
        double abs_int, abs_int_ref, xdiff, ydiff, ydiff_ref, H_real, H_imag, H_real_ref, H_imag_ref, val;

        // Get data
        xdiff       = x.getDeviceValue(r) - x_int.getDeviceValue(c);
        ydiff       = y.getDeviceValue(r) - y_int.getDeviceValue(c);
        ydiff_ref   = y.getDeviceValue(r) + y_int.getDeviceValue(c);
        abs_int     = std::sqrt(xdiff * xdiff + ydiff     * ydiff);
        abs_int_ref = std::sqrt(xdiff * xdiff + ydiff_ref * ydiff_ref);

        // Compute second Hankel functions
        H_real     = H12_real(k0 * abs_int);
        H_real_ref = H12_real(k0 * abs_int_ref);
        H_imag     = H12_imag(k0 * abs_int);
        H_imag_ref = H12_imag(k0 * abs_int_ref);

        val = constant * (1/abs_int      * H_imag     * ydiff + \
                Gamma_ref * 1/abs_int_ref * H_imag_ref * ydiff_ref);
        F2.setDeviceRealValue(r, c, val);
        val = -constant * (1/abs_int     * H_real     * ydiff + \
                Gamma_ref * 1/abs_int_ref * H_real_ref * ydiff_ref);
        F2.setDeviceImagValue(r, c, val);

        val = -constant * xdiff * (1/abs_int      * H_imag      + \
                        Gamma_ref * 1/abs_int_ref  * H_imag_ref);
        F3.setDeviceRealValue(r, c, val);
        val = constant * xdiff * (1/abs_int     * H_real      + \
                    Gamma_ref * 1/abs_int_ref * H_real_ref);
        F3.setDeviceImagValue(r, c, val);
        //if (r == 0 && c == 0) printf("hej fra kernel\n");
        
    }
}

void Segment::computeScatteredFieldMatrices(RealMatrix x, RealMatrix y, bool far_field_approximation) {
    
    int rows = y.rows;
    int cols = y_int.rows;

    ComplexMatrix F1, F2, F3;
    double constant, k0, Gamma_ref;

    k0 = constants.k0;
    Gamma_ref = constants.Gamma_ref;

    if (polarisation == 1) {
        F1 = E_scat_matrix.z;
        F2 = H_scat_matrix.x;
        F3 = H_scat_matrix.y;
        constant = 1/constants.eta0;
    }
    else if (polarisation == 2) {
        F1 = H_scat_matrix.z;
        F2 = E_scat_matrix.x;
        F3 = E_scat_matrix.y;
        constant = -constants.mu0/(constants.eta0*constants.epsilon0);
    } 
    else {
        printf("Please input 1 or 2 for the polarisation!\n");
        return;
    }
    if (deviceComputation) {

        // Blocks and threads
        dim3 dimBlock(32,16);
        dim3 dimGrid((rows + dimBlock.x - 1)/dimBlock.x, (cols + dimBlock.y - 1)/dimBlock.y);
        
        computeScatteredFieldMatricesKernelH02<<<dimGrid, dimBlock>>>(F1, x, x_int, y, y_int, constant, rows, cols, k0, Gamma_ref);
        cudaCheckError();
        hipDeviceSynchronize();
        computeScatteredFieldMatricesKernelH12<<<dimGrid, dimBlock>>>(F2, F3, x, x_int, y, y_int, constant, rows, cols, k0, Gamma_ref);
        cudaCheckError();
        hipDeviceSynchronize();
        
    }
    else {
        #pragma omp parallel for collapse(2) 
        for (int r = 0; r < rows; r++) {
            for (int c = 0; c < cols; c++) {
                double abs_int, abs_int_ref, xdiff, ydiff, ydiff_ref, H_real, H_imag, H_real_ref, H_imag_ref, val;

                // Get data
                xdiff       = x.getHostValue(r) - x_int.getHostValue(c);
                ydiff       = y.getHostValue(r) - y_int.getHostValue(c);
                ydiff_ref   = y.getHostValue(r) + y_int.getHostValue(c);
                abs_int     = std::sqrt(xdiff * xdiff + ydiff     * ydiff);
                abs_int_ref = std::sqrt(xdiff * xdiff + ydiff_ref * ydiff_ref);

                // Compute first Hankel functions
                H_real     = H02_real(k0 * abs_int);
                H_real_ref = H02_real(k0 * abs_int_ref);
                H_imag     = H02_imag(k0 * abs_int);
                H_imag_ref = H02_imag(k0 * abs_int_ref);
                
                val = H_real + Gamma_ref * H_real_ref;
                F1.setHostRealValue(r, c, val);
                val = H_imag + Gamma_ref * H_imag_ref;
                F1.setHostImagValue(r, c, val);

                // Compute second Hankel functions
                H_real     = H12_real(k0 * abs_int);
                H_real_ref = H12_real(k0 * abs_int_ref);
                H_imag     = H12_imag(k0 * abs_int);
                H_imag_ref = H12_imag(k0 * abs_int_ref);

                val = constant * (1/abs_int      * H_imag     * ydiff + \
                        Gamma_ref * 1/abs_int_ref * H_imag_ref * ydiff_ref);
                F2.setHostRealValue(r, c, val);
                val = -constant * (1/abs_int     * H_real     * ydiff + \
                        Gamma_ref * 1/abs_int_ref * H_real_ref * ydiff_ref);
                F2.setHostImagValue(r, c, val);

                val = -constant * xdiff * (1/abs_int      * H_imag      + \
                                Gamma_ref * 1/abs_int_ref  * H_imag_ref);
                F3.setHostRealValue(r, c, val);
                val = constant * xdiff * (1/abs_int     * H_real      + \
                            Gamma_ref * 1/abs_int_ref * H_real_ref);
                F3.setHostImagValue(r, c, val);
            }
        }
    }
}

__global__ void computeInteriorFieldMatricesKernel(ComplexMatrix F1, ComplexMatrix F2, ComplexMatrix F3, RealMatrix x, RealMatrix x_ext, RealMatrix y, RealMatrix y_ext, double constant, int rows, int cols, double k1) {
    int r = threadIdx.x + blockIdx.x * blockDim.x;
    int c = threadIdx.y + blockIdx.y * blockDim.y;
    if (r < rows && c < cols) {
        double abs_ext, xdiff, ydiff, H_real, H_imag, val;

        // Get data
        xdiff   = x.getDeviceValue(r) - x_ext.getDeviceValue(c);
        ydiff   = y.getDeviceValue(r) - y_ext.getDeviceValue(c);
        abs_ext = std::sqrt(xdiff*xdiff + ydiff*ydiff);

        // Compute first Hankel functions
        H_real = H02_real(k1 * abs_ext);
        H_imag = H02_imag(k1 * abs_ext);
        
        val = H_real;
        F1.setDeviceRealValue(r, c, val);
        val = H_imag;
        F1.setDeviceImagValue(r, c, val);

        // Compute second Hankel functions
        H_real = H12_real(k1 * abs_ext);
        H_imag = H12_imag(k1 * abs_ext);

        val =   constant * 1/abs_ext * ydiff * H_imag;
        F2.setDeviceRealValue(r, c, val);
        val = - constant * 1/abs_ext * ydiff * H_real;
        F2.setDeviceImagValue(r, c, val);

        val = -constant * 1/abs_ext * xdiff * H_imag;
        F3.setDeviceRealValue(r, c, val);
        val =  constant * 1/abs_ext * xdiff * H_real;
        F3.setDeviceImagValue(r, c, val);
    }
}

void Segment::computeInteriorFieldMatrices(RealMatrix x, RealMatrix y) {
    
    int rows = y.rows;
    int cols = y_ext.rows;

    ComplexMatrix F1, F2, F3;
    double constant, k1;

    k1 = constants.k1;

    if (polarisation == 1) {
        F1 = E_int_matrix.z;
        F2 = H_int_matrix.x;
        F3 = H_int_matrix.y;
        constant = constants.n1/constants.eta0;
    }
    else if (polarisation == 2) {
        F1 = H_int_matrix.z;
        F2 = E_int_matrix.x;
        F3 = E_int_matrix.y;
        constant = -constants.n1*constants.mu0/(constants.eta0*constants.epsilon0);
    } 
    else {
        printf("Please input 1 or 2 for the polarisation!\n");
    }

    if (true) {
        // Blocks and threads
        dim3 dimBlock(32,16);
        dim3 dimGrid((rows + dimBlock.x - 1)/dimBlock.x, (cols + dimBlock.y - 1)/dimBlock.y);
        
        computeInteriorFieldMatricesKernel<<<dimGrid, dimBlock>>>(F1, F2, F3, x, x_ext, y, y_ext, constant, rows, cols, k1);
        cudaCheckError();
        hipDeviceSynchronize();

    }
    else {
        #pragma omp parallel for collapse(2) 
        for (int r = 0; r < rows; r++) {
            for (int c = 0; c < cols; c++) {
                double abs_ext, xdiff, ydiff, H_real, H_imag, val;

                // Get data
                xdiff   = x.getHostValue(r) - x_ext.getHostValue(c);
                ydiff   = y.getHostValue(r) - y_ext.getHostValue(c);
                abs_ext = std::sqrt(xdiff*xdiff + ydiff*ydiff);

                // Compute first Hankel functions
                H_real = H02_real(k1 * abs_ext);
                H_imag = H02_imag(k1 * abs_ext);
                
                val = H_real;
                F1.setHostRealValue(r, c, val);
                val = H_imag;
                F1.setHostImagValue(r, c, val);

                // Compute second Hankel functions
                H_real = H12_real(k1 * abs_ext);
                H_imag = H12_imag(k1 * abs_ext);

                val =   constant * 1/abs_ext * ydiff * H_imag;
                F2.setHostRealValue(r, c, val);
                val = - constant * 1/abs_ext * ydiff * H_real;
                F2.setHostImagValue(r, c, val);

                val = -constant * 1/abs_ext * xdiff * H_imag;
                F3.setHostRealValue(r, c, val);
                val =  constant * 1/abs_ext * xdiff * H_real;
                F3.setHostImagValue(r, c, val);
            }
        }
    }
}


}