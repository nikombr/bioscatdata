#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <string.h>
extern "C" {
#include "../../../lib/RealMatrix.h"
using namespace std;

__global__ void computeInteriorPointsKernel(RealMatrix x_int, RealMatrix y_int, RealMatrix x_test, RealMatrix y_test, double alpha, int n_top, int n_right, int n_bottom, int n_left, int n) {
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    if (j < n) {
        double xdiff, ydiff, norm;
        int shift;
        
        if (j < n_top - 4) {
            shift = -2;
        }
        else if (j < n_top + n_right - 8) {
            shift = -6;
        }
        else if (j < n_top + n_right + n_bottom - 12) {
            shift = -10;
        }
        else {
            shift = -14;
        }
        xdiff = x_test.getDeviceValue(j - 1 - shift) - x_test.getDeviceValue(j + 1 - shift); // Central difference
        ydiff = y_test.getDeviceValue(j - 1 - shift) - y_test.getDeviceValue(j + 1 - shift); // Central difference
        
        norm = std::sqrt(xdiff*xdiff + ydiff*ydiff);
        xdiff /= norm;
        ydiff /= norm;
        //printf("alpha=%e\n",alpha);
        x_int.setDeviceValue(j, x_test.getDeviceValue(j - shift) - alpha*ydiff);
        y_int.setDeviceValue(j, y_test.getDeviceValue(j - shift) + alpha*xdiff);
    }
}

void computeInteriorPoints(RealMatrix x_int, RealMatrix y_int, RealMatrix x_test, RealMatrix y_test, double alpha, int n_top, int n_right, int n_bottom, int n_left, bool deviceComputation, bool printOutput) {
    int n = x_int.rows;
    if (deviceComputation) { // GPU
        if (printOutput) printf("Computing interior points on the GPU.\n");
        // Blocks and threads
        dim3 dimBlock(256);
        dim3 dimGrid((n + dimBlock.x - 1)/dimBlock.x);
        computeInteriorPointsKernel<<<dimGrid, dimBlock>>>(x_int, y_int, x_test, y_test, alpha, n_top, n_right, n_bottom, n_left, n); 
        hipDeviceSynchronize();
    }
    else { // CPU
        if (printOutput) printf("Computing interior points on the CPU.\n");
        for (int j = 0; j < n; j++) {
            double xdiff, ydiff, norm;
            int shift;
            
            if (j < n_top - 4) {
                shift = -2;
            }
            else if (j < n_top + n_right - 8) {
                shift = -6;
            }
            else if (j < n_top + n_right + n_bottom - 12) {
                shift = -10;
            }
            else {
                shift = -14;
            }
            xdiff = x_test.getHostValue(j - 1 - shift) - x_test.getHostValue(j + 1 - shift); // Central difference
            ydiff = y_test.getHostValue(j - 1 - shift) - y_test.getHostValue(j + 1 - shift); // Central difference
            
            norm = std::sqrt(xdiff*xdiff + ydiff*ydiff);
            xdiff /= norm;
            ydiff /= norm;
            //printf("alpha=%e\n",alpha);
            x_int.setHostValue(j, x_test.getHostValue(j - shift) - alpha*ydiff);
            y_int.setHostValue(j, y_test.getHostValue(j - shift) + alpha*xdiff);

        }

    }
}


}