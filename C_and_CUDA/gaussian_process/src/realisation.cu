#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hipblas.h>
#include <stdio.h>
#include <math.h>

extern "C" {
#include "../lib/GaussianProcess.h"
#include <cblas.h>
#include <ctime>


void GaussianProcess::generate_random_vector() {

    double pi = 3.14159265358979323846;
    double U1, U2;

    for (int i = 0; i < n; i+=2) {
        U1 = ((double) rand())/((double) RAND_MAX);
        U2 = ((double) rand())/((double) RAND_MAX);
        p_h[i]   = sqrt(-2*log(U1))*cos(2*pi*U2);
        p_h[i+1] = sqrt(-2*log(U1))*sin(2*pi*U2);

    }

    if (n < 21) {
        printf("\n");
        for (int k = 0; k < n; k++) {
            if (k != n-1) printf("%.8f, ",p_h[k]);
            else printf("%.8f",p_h[k]);
            
        }
        printf("\n");
    }

    if (device) {
        // Send to device
        hipMemcpy(p_d, p_h, n * sizeof(double), hipMemcpyHostToDevice);
    }


}

void GaussianProcess::realisation() {

    generate_random_vector();

    if (device) {

        // cuBLAS handle creation
        hipblasHandle_t handle;
        hipblasStatus_t status;
        status = hipblasCreate(&handle);

        if (status != HIPBLAS_STATUS_SUCCESS) {
            printf("cuBLAS initialization failed %d\n",status);
            return;
        }
        
        status = hipblasDtrmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT, n, M_log, n, p_d, 1);
        
        // Check if hipblasDtrmv was successful
        if (status != HIPBLAS_STATUS_SUCCESS) {
            printf("hipblasDtrmv failed with error code: %d\n", status);
        }

        // Send to host
        hipMemcpy(p_h, p_d, n * sizeof(double), hipMemcpyDeviceToHost);

        // Destroy cuBLAS handle
        status = hipblasDestroy(handle);
        if (status != HIPBLAS_STATUS_SUCCESS) {
            printf("CUBLAS destruction failed\n");
            return;
        }
    }
    else {
        cblas_dtrmv(CblasRowMajor, CblasLower, CblasNoTrans, CblasNonUnit, n, *M_h, n, p_h, 1);
    }

}


}